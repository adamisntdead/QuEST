#include "hip/hip_runtime.h"
/** @file
An implementation of the API in qubits.h for a local (non-MPI) environment.
*/

# include <stdlib.h>
# include <stdio.h>
# include <omp.h>
# include "qubits.h"
# include "qubits_internal.h"

void createMultiQubit(MultiQubit *multiQubit, int numQubits, QUESTEnv env)
{
	createMultiQubitCPU(multiQubit, numQubits, env);
	hipMalloc(&(multiQubit->deviceStateVec.real), multiQubit->numAmps*sizeof(multiQubit->deviceStateVec.real));
	hipMalloc(&(multiQubit->deviceStateVec.imag), multiQubit->numAmps*sizeof(multiQubit->deviceStateVec.imag));

        if (!(multiQubit->deviceStateVec.real) || !(multiQubit->deviceStateVec.imag)){
                printf("Could not allocate memory on GPU!\n");
                exit (EXIT_FAILURE);
        }

}

void destroyMultiQubit(MultiQubit multiQubit, QUESTEnv env)
{
	destroyMultiQubitCPU(multiQubit, env);
	hipFree(multiQubit.deviceStateVec.real);
	hipFree(multiQubit.deviceStateVec.imag);
}

int GPUExists(void){
	int deviceCount, device;
	int gpuDeviceCount = 0;
	struct hipDeviceProp_t properties;
	hipError_t cudaResultCode = hipGetDeviceCount(&deviceCount);
	if (cudaResultCode != hipSuccess) deviceCount = 0;
	/* machines with no GPUs can still report one emulation device */
	for (device = 0; device < deviceCount; ++device) {
		hipGetDeviceProperties(&properties, device);
		if (properties.major != 9999) { /* 9999 means emulation only */
			++gpuDeviceCount;
		}
	}
	if (gpuDeviceCount) return 1;
	else return 0;
}

void initQUESTEnv(QUESTEnv *env){
        // init MPI environment
	if (!GPUExists()){
		printf("Trying to run GPU code with no GPU available\n");
		exit(EXIT_FAILURE);
	}
	env->rank=0;
	env->numRanks=1;
}

void syncQUESTEnv(QUESTEnv env){
	hipDeviceSynchronize();
} 

void closeQUESTEnv(QUESTEnv env){
	// MPI finalize goes here in MPI version. Call this function anyway for consistency
}

void reportQUESTEnv(QUESTEnv env){
	printf("EXECUTION ENVIRONMENT:\n");
	printf("Running locally on one node with GPU\n");
	printf("Number of ranks is %d\n", env.numRanks);
# ifdef _OPENMP
	printf("OpenMP enabled\n");
	printf("Number of threads available is %d\n", omp_get_max_threads());
# else
	printf("OpenMP disabled\n");
# endif
}

void copyStateToGPU(MultiQubit multiQubit)
{
	printf("Copying data to GPU\n");
        hipMemcpy(multiQubit.deviceStateVec.real, multiQubit.stateVec.real, 
			multiQubit.numAmps*sizeof(multiQubit.deviceStateVec.real), hipMemcpyHostToDevice);
        hipMemcpy(multiQubit.deviceStateVec.imag, multiQubit.stateVec.imag, 
			multiQubit.numAmps*sizeof(multiQubit.deviceStateVec.imag), hipMemcpyHostToDevice);
	printf("Finished copying data to GPU\n");
}

void copyStateFromGPU(MultiQubit multiQubit)
{
	hipDeviceSynchronize();
	printf("Copying data from GPU\n");
        hipMemcpy(multiQubit.stateVec.real, multiQubit.deviceStateVec.real, 
			multiQubit.numAmps*sizeof(multiQubit.deviceStateVec.real), hipMemcpyDeviceToHost);
        hipMemcpy(multiQubit.stateVec.imag, multiQubit.deviceStateVec.imag, 
			multiQubit.numAmps*sizeof(multiQubit.deviceStateVec.imag), hipMemcpyDeviceToHost);
	printf("Finished copying data from GPU\n");
}


void initStateVec(MultiQubit *multiQubit)
{
	initStateVecCPU(multiQubit);
	copyStateToGPU(*multiQubit);
}

double calcTotalProbability(MultiQubit multiQubit){
        double pTotal=0; 
	long long int index;
	long long int numAmpsPerRank = multiQubit.numAmps;

	copyStateFromGPU(multiQubit);

        for (index=0; index<numAmpsPerRank; index++){ 
                pTotal+=multiQubit.stateVec.real[index]*multiQubit.stateVec.real[index];      
                pTotal+=multiQubit.stateVec.imag[index]*multiQubit.stateVec.imag[index];      
        } 
	return pTotal;
}


__global__ void rotateQubitKernel (MultiQubit multiQubit, const int rotQubit, Complex alpha, Complex beta){
// ----- sizes
        long long int sizeBlock,                                           // size of blocks
        sizeHalfBlock;                                       // size of blocks halved
        // ----- indices
        long long int thisBlock,                                           // current block
             indexUp,indexLo;                                     // current index and corresponding index in lower half block

        // ----- temp variables
        double   stateRealUp,stateRealLo,                             // storage for previous state values
                 stateImagUp,stateImagLo;                             // (used in updates)
        // ----- temp variables
        long long int thisTask;                                   // task based approach for expose loop with small granularity
        const long long int numTasks=multiQubit.numAmps>>1;
        // (good for shared memory parallelism)


        // ---------------------------------------------------------------- //
        //            tests                                                 //
        // ---------------------------------------------------------------- //
        //assert (rotQubit >= 0 && rotQubit < multiQubit.numQubits);


        // ---------------------------------------------------------------- //
        //            dimensions                                            //
        // ---------------------------------------------------------------- //
        sizeHalfBlock = 1LL << rotQubit;                               // size of blocks halved
        sizeBlock     = 2LL * sizeHalfBlock;                           // size of blocks


        // ---------------------------------------------------------------- //
        //            rotate                                                //
        // ---------------------------------------------------------------- //

        //
        // --- task-based shared-memory parallel implementation
        //

        // Can't use multiQubit.stateVec as a private OMP var
	//! fix -- no necessary for GPU version
        double *stateVecReal = multiQubit.deviceStateVec.real;
        double *stateVecImag = multiQubit.deviceStateVec.imag;
        double alphaImag=alpha.imag, alphaReal=alpha.real;
        double betaImag=beta.imag, betaReal=beta.real;

	thisTask = blockIdx.x*blockDim.x + threadIdx.x;
	if (thisTask>=numTasks) return;

	thisBlock   = thisTask / sizeHalfBlock;
	indexUp     = thisBlock*sizeBlock + thisTask%sizeHalfBlock;
	indexLo     = indexUp + sizeHalfBlock;

	// store current state vector values in temp variables
	stateRealUp = stateVecReal[indexUp];
	stateImagUp = stateVecImag[indexUp];

	stateRealLo = stateVecReal[indexLo];
	stateImagLo = stateVecImag[indexLo];

	// state[indexUp] = alpha * state[indexUp] - conj(beta)  * state[indexLo]
	stateVecReal[indexUp] = alphaReal*stateRealUp - alphaImag*stateImagUp 
		- betaReal*stateRealLo - betaImag*stateImagLo;
	stateVecImag[indexUp] = alphaReal*stateImagUp + alphaImag*stateRealUp 
		- betaReal*stateImagLo + betaImag*stateRealLo;

	// state[indexLo] = beta  * state[indexUp] + conj(alpha) * state[indexLo]
	stateVecReal[indexLo] = betaReal*stateRealUp - betaImag*stateImagUp 
		+ alphaReal*stateRealLo + alphaImag*stateImagLo;
	stateVecImag[indexLo] = betaReal*stateImagUp + betaImag*stateRealUp 
		+ alphaReal*stateImagLo - alphaImag*stateRealLo;
}

void rotateQubit(MultiQubit multiQubit, const int rotQubit, Complex alpha, Complex beta) 
{
        int threadsPerCUDABlock, CUDABlocks;

        threadsPerCUDABlock = 128;
        CUDABlocks = ceil((double)(multiQubit.numAmps>>1)/threadsPerCUDABlock);
        //printf("cuda blocks: %d\n", CUDABlocks);

        rotateQubitKernel<<<CUDABlocks, threadsPerCUDABlock>>>(multiQubit, rotQubit, alpha, beta);
}

double findProbabilityOfZero(MultiQubit multiQubit,
                const int measureQubit)
{
	double stateProb=0;
	stateProb = findProbabilityOfZeroLocal(multiQubit, measureQubit);
	return stateProb;
}

double measureInZero(MultiQubit multiQubit, const int measureQubit)
{
        double stateProb;
	stateProb = findProbabilityOfZero(multiQubit, measureQubit);
        measureInZeroLocal(multiQubit, measureQubit, stateProb);
        return stateProb;
}

double filterOut111(MultiQubit multiQubit, const int idQubit1, const int idQubit2, const int idQubit3)
{
        double stateProb=0;
        stateProb = probOfFilterOut111(multiQubit, idQubit1, idQubit2, idQubit3);
        filterOut111Local(multiQubit, idQubit1, idQubit2, idQubit3, stateProb);
        return stateProb;
}

double probOfFilterOut111(MultiQubit multiQubit, const int idQubit1, const int idQubit2, const int idQubit3)
{
        double stateProb=0;
        stateProb = probOfFilterOut111Local(multiQubit, idQubit1, idQubit2, idQubit3);
        return stateProb;
}



