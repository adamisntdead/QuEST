#include "hip/hip_runtime.h"
/** @file
An implementation of the API in qubits.h for a local (non-MPI) environment.
*/

# include <stdlib.h>
# include <stdio.h>
# include <omp.h>
# include "qubits.h"
# include "qubits_internal.h"

# define REDUCE_SHARED_SIZE 512

static __device__ int extractBit (int locationOfBitFromRight, long long int theEncodedNumber)
{
        return (theEncodedNumber & ( 1LL << locationOfBitFromRight )) >> locationOfBitFromRight;
}

void createMultiQubit(MultiQubit *multiQubit, int numQubits, QUESTEnv env)
{
	createMultiQubitCPU(multiQubit, numQubits, env);
	hipMalloc(&(multiQubit->deviceStateVec.real), multiQubit->numAmps*sizeof(multiQubit->deviceStateVec.real));
	hipMalloc(&(multiQubit->deviceStateVec.imag), multiQubit->numAmps*sizeof(multiQubit->deviceStateVec.imag));
	hipMalloc(&(multiQubit->firstLevelReduction), ceil(multiQubit->numAmps/(double)REDUCE_SHARED_SIZE)*sizeof(double));
	hipMalloc(&(multiQubit->secondLevelReduction), ceil(multiQubit->numAmps/(double)(REDUCE_SHARED_SIZE*REDUCE_SHARED_SIZE))*
		sizeof(double));

        if (!(multiQubit->deviceStateVec.real) || !(multiQubit->deviceStateVec.imag)){
                printf("Could not allocate memory on GPU!\n");
                exit (EXIT_FAILURE);
        }

}

void destroyMultiQubit(MultiQubit multiQubit, QUESTEnv env)
{
	destroyMultiQubitCPU(multiQubit, env);
	hipFree(multiQubit.deviceStateVec.real);
	hipFree(multiQubit.deviceStateVec.imag);
}

int GPUExists(void){
	int deviceCount, device;
	int gpuDeviceCount = 0;
	struct hipDeviceProp_t properties;
	hipError_t cudaResultCode = hipGetDeviceCount(&deviceCount);
	if (cudaResultCode != hipSuccess) deviceCount = 0;
	/* machines with no GPUs can still report one emulation device */
	for (device = 0; device < deviceCount; ++device) {
		hipGetDeviceProperties(&properties, device);
		if (properties.major != 9999) { /* 9999 means emulation only */
			++gpuDeviceCount;
		}
	}
	if (gpuDeviceCount) return 1;
	else return 0;
}

void initQUESTEnv(QUESTEnv *env){
        // init MPI environment
	if (!GPUExists()){
		printf("Trying to run GPU code with no GPU available\n");
		exit(EXIT_FAILURE);
	}
	env->rank=0;
	env->numRanks=1;
}

void syncQUESTEnv(QUESTEnv env){
	hipDeviceSynchronize();
} 

void closeQUESTEnv(QUESTEnv env){
	// MPI finalize goes here in MPI version. Call this function anyway for consistency
}

void reportQUESTEnv(QUESTEnv env){
	printf("EXECUTION ENVIRONMENT:\n");
	printf("Running locally on one node with GPU\n");
	printf("Number of ranks is %d\n", env.numRanks);
# ifdef _OPENMP
	printf("OpenMP enabled\n");
	printf("Number of threads available is %d\n", omp_get_max_threads());
# else
	printf("OpenMP disabled\n");
# endif
}

void copyStateToGPU(MultiQubit multiQubit)
{
	printf("Copying data to GPU\n");
        hipMemcpy(multiQubit.deviceStateVec.real, multiQubit.stateVec.real, 
			multiQubit.numAmps*sizeof(multiQubit.deviceStateVec.real), hipMemcpyHostToDevice);
        hipMemcpy(multiQubit.deviceStateVec.imag, multiQubit.stateVec.imag, 
			multiQubit.numAmps*sizeof(multiQubit.deviceStateVec.imag), hipMemcpyHostToDevice);
	printf("Finished copying data to GPU\n");
}

void copyStateFromGPU(MultiQubit multiQubit)
{
	hipDeviceSynchronize();
	printf("Copying data from GPU\n");
        hipMemcpy(multiQubit.stateVec.real, multiQubit.deviceStateVec.real, 
			multiQubit.numAmps*sizeof(multiQubit.deviceStateVec.real), hipMemcpyDeviceToHost);
        hipMemcpy(multiQubit.stateVec.imag, multiQubit.deviceStateVec.imag, 
			multiQubit.numAmps*sizeof(multiQubit.deviceStateVec.imag), hipMemcpyDeviceToHost);
	printf("Finished copying data from GPU\n");
}


void initStateVec(MultiQubit *multiQubit)
{
	initStateVecCPU(multiQubit);
	copyStateToGPU(*multiQubit);
}

double calcTotalProbability(MultiQubit multiQubit){
        double pTotal=0; 
	long long int index;
	long long int numAmpsPerRank = multiQubit.numAmps;

	copyStateFromGPU(multiQubit);

        for (index=0; index<numAmpsPerRank; index++){ 
                pTotal+=multiQubit.stateVec.real[index]*multiQubit.stateVec.real[index];      
                pTotal+=multiQubit.stateVec.imag[index]*multiQubit.stateVec.imag[index];      
        } 
	return pTotal;
}


__global__ void rotateQubitKernel (MultiQubit multiQubit, const int rotQubit, Complex alpha, Complex beta){
// ----- sizes
        long long int sizeBlock,                                           // size of blocks
        sizeHalfBlock;                                       // size of blocks halved
        // ----- indices
        long long int thisBlock,                                           // current block
             indexUp,indexLo;                                     // current index and corresponding index in lower half block

        // ----- temp variables
        double   stateRealUp,stateRealLo,                             // storage for previous state values
                 stateImagUp,stateImagLo;                             // (used in updates)
        // ----- temp variables
        long long int thisTask;                                   // task based approach for expose loop with small granularity
        const long long int numTasks=multiQubit.numAmps>>1;
        // (good for shared memory parallelism)


        // ---------------------------------------------------------------- //
        //            tests                                                 //
        // ---------------------------------------------------------------- //
        //assert (rotQubit >= 0 && rotQubit < multiQubit.numQubits);


        // ---------------------------------------------------------------- //
        //            dimensions                                            //
        // ---------------------------------------------------------------- //
        sizeHalfBlock = 1LL << rotQubit;                               // size of blocks halved
        sizeBlock     = 2LL * sizeHalfBlock;                           // size of blocks


        // ---------------------------------------------------------------- //
        //            rotate                                                //
        // ---------------------------------------------------------------- //

        //
        // --- task-based shared-memory parallel implementation
        //

        // Can't use multiQubit.stateVec as a private OMP var
	//! fix -- no necessary for GPU version
        double *stateVecReal = multiQubit.deviceStateVec.real;
        double *stateVecImag = multiQubit.deviceStateVec.imag;
        double alphaImag=alpha.imag, alphaReal=alpha.real;
        double betaImag=beta.imag, betaReal=beta.real;

	thisTask = blockIdx.x*blockDim.x + threadIdx.x;
	if (thisTask>=numTasks) return;

	thisBlock   = thisTask / sizeHalfBlock;
	indexUp     = thisBlock*sizeBlock + thisTask%sizeHalfBlock;
	indexLo     = indexUp + sizeHalfBlock;

	// store current state vector values in temp variables
	stateRealUp = stateVecReal[indexUp];
	stateImagUp = stateVecImag[indexUp];

	stateRealLo = stateVecReal[indexLo];
	stateImagLo = stateVecImag[indexLo];

	// state[indexUp] = alpha * state[indexUp] - conj(beta)  * state[indexLo]
	stateVecReal[indexUp] = alphaReal*stateRealUp - alphaImag*stateImagUp 
		- betaReal*stateRealLo - betaImag*stateImagLo;
	stateVecImag[indexUp] = alphaReal*stateImagUp + alphaImag*stateRealUp 
		- betaReal*stateImagLo + betaImag*stateRealLo;

	// state[indexLo] = beta  * state[indexUp] + conj(alpha) * state[indexLo]
	stateVecReal[indexLo] = betaReal*stateRealUp - betaImag*stateImagUp 
		+ alphaReal*stateRealLo + alphaImag*stateImagLo;
	stateVecImag[indexLo] = betaReal*stateImagUp + betaImag*stateRealUp 
		+ alphaReal*stateImagLo - alphaImag*stateRealLo;
}

void rotateQubit(MultiQubit multiQubit, const int rotQubit, Complex alpha, Complex beta) 
{
        int threadsPerCUDABlock, CUDABlocks;
        threadsPerCUDABlock = 128;
        CUDABlocks = ceil((double)(multiQubit.numAmps>>1)/threadsPerCUDABlock);
        rotateQubitKernel<<<CUDABlocks, threadsPerCUDABlock>>>(multiQubit, rotQubit, alpha, beta);
}


__global__ void controlPhaseGateKernel(MultiQubit multiQubit, const int idQubit1, const int idQubit2)
{
        long long int index;
        long long int stateVecSize;
        int bit1, bit2;

        stateVecSize = multiQubit.numAmps;
        double *stateVecReal = multiQubit.deviceStateVec.real;
        double *stateVecImag = multiQubit.deviceStateVec.imag;

	index = blockIdx.x*blockDim.x + threadIdx.x;
	if (index>=stateVecSize) return;

	bit1 = extractBit (idQubit1, index);
	bit2 = extractBit (idQubit2, index);
	if (bit1 && bit2) {
		stateVecReal [index] = - stateVecReal [index];
		stateVecImag [index] = - stateVecImag [index];
	}
}

void controlPhaseGate(MultiQubit multiQubit, const int idQubit1, const int idQubit2)
{
        int threadsPerCUDABlock, CUDABlocks;
        threadsPerCUDABlock = 128;
        CUDABlocks = ceil((double)(multiQubit.numAmps)/threadsPerCUDABlock);
        controlPhaseGateKernel<<<CUDABlocks, threadsPerCUDABlock>>>(multiQubit, idQubit1, idQubit2);
}

__global__ void quadCPhaseGateKernel(MultiQubit multiQubit, const int idQubit1, const int idQubit2, 
                const int idQubit3, const int idQubit4)
{
        long long int index;
        long long int stateVecSize;
        int bit1, bit2, bit3, bit4;

        stateVecSize = multiQubit.numAmps;
        double *stateVecReal = multiQubit.deviceStateVec.real;
        double *stateVecImag = multiQubit.deviceStateVec.imag;
	
	index = blockIdx.x*blockDim.x + threadIdx.x;
	if (index>=stateVecSize) return;
	
	bit1 = extractBit (idQubit1, index);
	bit2 = extractBit (idQubit2, index);
	bit3 = extractBit (idQubit3, index);
	bit4 = extractBit (idQubit4, index);
	if (bit1 && bit2 && bit3 && bit4) {
		stateVecReal [index] = - stateVecReal [index];
		stateVecImag [index] = - stateVecImag [index];
	}
}

void quadCPhaseGate(MultiQubit multiQubit, const int idQubit1, const int idQubit2,
		const int idQubit3, const int idQubit4)
{
        int threadsPerCUDABlock, CUDABlocks;
        threadsPerCUDABlock = 128;
        CUDABlocks = ceil((double)(multiQubit.numAmps)/threadsPerCUDABlock);
        quadCPhaseGateKernel<<<CUDABlocks, threadsPerCUDABlock>>>(multiQubit, idQubit1, idQubit2, idQubit3, idQubit4);
}

__device__ __host__ unsigned int log2Int( unsigned int x )
{
        unsigned int ans = 0 ;
        while( x>>=1 ) ans++;
        return ans ;
}

__device__ void reduceBlock(double *arrayIn, double *reducedArray, int length){
        int i, l, r;
        int threadMax, maxDepth;
        threadMax = length/2;
	maxDepth = log2Int(length/2);

        for (i=0; i<maxDepth+1; i++){
                if (threadIdx.x<threadMax){
                        l = threadIdx.x;
                        r = l + threadMax;
                        arrayIn[l] = arrayIn[r] + arrayIn[l];
                }
                threadMax = threadMax >> 1;
                __syncthreads(); // optimise -- use warp shuffle instead
        }

        if (threadIdx.x==0) reducedArray[blockIdx.x] = arrayIn[0];
}

__global__ void copySharedReduceBlock(double*arrayIn, double *reducedArray, int length){
	extern __shared__ double tempReductionArray[];
	int blockOffset = blockIdx.x*length;
	tempReductionArray[threadIdx.x*2] = arrayIn[blockOffset + threadIdx.x*2];
	tempReductionArray[threadIdx.x*2+1] = arrayIn[blockOffset + threadIdx.x*2+1];
	__syncthreads();
	reduceBlock(tempReductionArray, reducedArray, length);
}

__global__ void findProbabilityOfZeroKernel(MultiQubit multiQubit,
                const int measureQubit, double *reducedArray)
{
        // ----- sizes
        long long int sizeBlock,                                           // size of blocks
        sizeHalfBlock;                                       // size of blocks halved
        // ----- indices
        long long int thisBlock,                                           // current block
             index;                                               // current index for first half block
        // ----- temp variables
        long long int thisTask;                                   // task based approach for expose loop with small granularity
        long long int numTasks=multiQubit.numAmps>>1;
        // (good for shared memory parallelism)

	extern __shared__ double tempReductionArray[];

        // ---------------------------------------------------------------- //
        //            tests                                                 //
        // ---------------------------------------------------------------- //

        // ---------------------------------------------------------------- //
        //            dimensions                                            //
        // ---------------------------------------------------------------- //
        sizeHalfBlock = 1LL << (measureQubit);                       // number of state vector elements to sum,
        // and then the number to skip
        sizeBlock     = 2LL * sizeHalfBlock;                           // size of blocks (pairs of measure and skip entries)

        // ---------------------------------------------------------------- //
        //            find probability                                      //
        // ---------------------------------------------------------------- //

        //
        // --- task-based shared-memory parallel implementation
        //

        double *stateVecReal = multiQubit.deviceStateVec.real;
        double *stateVecImag = multiQubit.deviceStateVec.imag;

	thisTask = blockIdx.x*blockDim.x + threadIdx.x;
	if (thisTask>=numTasks) return;

	thisBlock = thisTask / sizeHalfBlock;
	index     = thisBlock*sizeBlock + thisTask%sizeHalfBlock;
	double realVal, imagVal;
	realVal = stateVecReal[index];
	imagVal = stateVecImag[index]; 	
	tempReductionArray[threadIdx.x] = realVal*realVal + imagVal*imagVal;
	__syncthreads();

	if (threadIdx.x<blockDim.x/2){
		reduceBlock(tempReductionArray, reducedArray, blockDim.x);
	}
}

int getNumReductionLevels(long long int numValuesToReduce, int numReducedPerLevel){
	int levels=0;
	while (numValuesToReduce){
		numValuesToReduce = numValuesToReduce/numReducedPerLevel;
		levels++;
	}
	return levels;
}

void swapDouble(double **a, double **b){
        double *temp;
        temp = *a;
        *a = *b;
        *b = temp;
}

double findProbabilityOfZero(MultiQubit multiQubit,
                const int measureQubit)
{
	long long int numValuesToReduce = multiQubit.numAmps>>1;
	int valuesPerCUDABlock, numCUDABlocks, sharedMemSize;
	double stateProb=0;
	int firstTime=1;
	int maxReducedPerLevel = REDUCE_SHARED_SIZE;

	while(numValuesToReduce>1){	
		if (numValuesToReduce<maxReducedPerLevel){
			// Need less than one CUDA block to reduce values
			valuesPerCUDABlock = numValuesToReduce;
			numCUDABlocks = 1;
		} else {
			// Use full CUDA blocks, with block size constrained by shared mem usage
			valuesPerCUDABlock = maxReducedPerLevel;
			numCUDABlocks = ceil((double)numValuesToReduce/valuesPerCUDABlock);
		}
		sharedMemSize = valuesPerCUDABlock*sizeof(double);

		if (firstTime){
			findProbabilityOfZeroKernel<<<numCUDABlocks, valuesPerCUDABlock, sharedMemSize>>>(
				multiQubit, measureQubit, multiQubit.firstLevelReduction);
			firstTime=0;
		} else {
			hipDeviceSynchronize();	
			copySharedReduceBlock<<<numCUDABlocks, valuesPerCUDABlock/2, sharedMemSize>>>(
				multiQubit.firstLevelReduction, 
				multiQubit.secondLevelReduction, valuesPerCUDABlock); 
			hipDeviceSynchronize();	
			swapDouble(&(multiQubit.firstLevelReduction), &(multiQubit.secondLevelReduction));
		}
		numValuesToReduce = numValuesToReduce/maxReducedPerLevel;
	}
	hipMemcpy(&stateProb, multiQubit.firstLevelReduction, sizeof(double), hipMemcpyDeviceToHost);
	return stateProb;
}


__global__ void measureInZeroKernel(MultiQubit multiQubit, int measureQubit, double totalProbability)
{
        // ----- sizes
        long long int sizeBlock,                                           // size of blocks
        sizeHalfBlock;                                       // size of blocks halved
        // ----- indices
        long long int thisBlock,                                           // current block
             index;                                               // current index for first half block
        // ----- measured probability
        double   renorm;                                    // probability (returned) value
        // ----- temp variables
        long long int thisTask;                                   // task based approach for expose loop with small granularity
        // (good for shared memory parallelism)
        long long int numTasks=multiQubit.numAmps>>1;

        // ---------------------------------------------------------------- //
        //            tests                                                 //
        // ---------------------------------------------------------------- //
        // ---------------------------------------------------------------- //
        //            dimensions                                            //
        // ---------------------------------------------------------------- //
        sizeHalfBlock = 1LL << (measureQubit);                       // number of state vector elements to sum,
        // and then the number to skip
        sizeBlock     = 2LL * sizeHalfBlock;                           // size of blocks (pairs of measure and skip entries)

        // ---------------------------------------------------------------- //
        //            find probability                                      //
        // ---------------------------------------------------------------- //

        //
        // --- task-based shared-memory parallel implementation
        //
        renorm=1/sqrt(totalProbability);
        double *stateVecReal = multiQubit.deviceStateVec.real;
        double *stateVecImag = multiQubit.deviceStateVec.imag;

	thisTask = blockIdx.x*blockDim.x + threadIdx.x;
	if (thisTask>=numTasks) return;
	thisBlock = thisTask / sizeHalfBlock;
	index     = thisBlock*sizeBlock + thisTask%sizeHalfBlock;
	stateVecReal[index]=stateVecReal[index]*renorm;
	stateVecImag[index]=stateVecImag[index]*renorm;

	stateVecReal[index+sizeHalfBlock]=0;
	stateVecImag[index+sizeHalfBlock]=0;
}

double measureInZero(MultiQubit multiQubit, const int measureQubit)
{        
        double stateProb;
	stateProb = findProbabilityOfZero(multiQubit, measureQubit);

	int threadsPerCUDABlock, CUDABlocks;
        threadsPerCUDABlock = 128;
        CUDABlocks = ceil((double)(multiQubit.numAmps>>1)/threadsPerCUDABlock);
        measureInZeroKernel<<<CUDABlocks, threadsPerCUDABlock>>>(multiQubit, measureQubit, stateProb);
        return stateProb;
}

/** Updates the state according to this scenario: we ask "are these 3 qubits in 111" and the answer is "no".
@param[in,out] multiQubit object representing the set of qubits
@param[in] idQubit1, idQubit2, idQubit3 specified qubits                 
@param[in] probOfFilter Total probability that the 3 qubits are not all in the 1 state. 
*/
__global__ void filterOut111Kernel(MultiQubit multiQubit, const int idQubit1, const int idQubit2, const int idQubit3,
                const double probOfFilter)
{
        long long int index;
        long long int stateVecSize;
        int bit1, bit2, bit3;

        stateVecSize = multiQubit.numAmps;

        double myNorm=1/sqrt(probOfFilter);
        double *stateVecReal = multiQubit.deviceStateVec.real;
        double *stateVecImag = multiQubit.deviceStateVec.imag;

	index = blockIdx.x*blockDim.x + threadIdx.x;
	if (index>=stateVecSize) return;
	bit1 = extractBit (idQubit1, index);
	bit2 = extractBit (idQubit2, index);
	bit3 = extractBit (idQubit3, index);
	if ((bit1 && bit2 && bit3)) {
		stateVecReal[index]=0;
		stateVecImag [index]=0;
	}else{
		stateVecReal[index] *= myNorm;
		stateVecImag[index] *= myNorm;
	}
}

double filterOut111(MultiQubit multiQubit, const int idQubit1, const int idQubit2, const int idQubit3)
{
        double stateProb=0;
        int threadsPerCUDABlock, CUDABlocks;
        threadsPerCUDABlock = 128;
        CUDABlocks = ceil((double)(multiQubit.numAmps)/threadsPerCUDABlock);

        stateProb = probOfFilterOut111(multiQubit, idQubit1, idQubit2, idQubit3);
        filterOut111Kernel<<<CUDABlocks, threadsPerCUDABlock>>>(multiQubit, idQubit1, idQubit2, idQubit3, stateProb);
        return stateProb;
}


/** Evaluates the state according to this scenario: we ask "are these 3 qubits in 111" and the answer is "no".
The function returns the probability of this outcome across all amplitudes in this chunk (if zero, it will exit with error) 
@param[in,out] multiQubit object representing the set of qubits
@param[in] idQubit1, idQubit2, idQubit3 specified qubits                 
@return Total probability that the 3 qubits are not all in the 1 state. 
*/
__global__ void probOfFilterOut111Kernel(MultiQubit multiQubit, const int idQubit1, const int idQubit2, const int idQubit3, double *reducedArray)
{
        long long int index;
        long long int stateVecSize;
        int bit1, bit2, bit3;

        stateVecSize = multiQubit.numAmps;

        double *stateVecReal = multiQubit.deviceStateVec.real;
        double *stateVecImag = multiQubit.deviceStateVec.imag;
	
	extern __shared__ double tempReductionArray[];

	index = blockIdx.x*blockDim.x + threadIdx.x;
	if (index>=stateVecSize) return;

	double realVal, imagVal;
	realVal = stateVecReal[index];
	imagVal = stateVecImag[index];

	bit1 = extractBit (idQubit1, index);
	bit2 = extractBit (idQubit2, index);
	bit3 = extractBit (idQubit3, index);
	if (!(bit1 && bit2 && bit3)) {
		tempReductionArray[threadIdx.x] = realVal*realVal + imagVal*imagVal;
	} else {
		tempReductionArray[threadIdx.x] = 0;
	}
	__syncthreads();
        
	if (threadIdx.x<blockDim.x/2){
		reduceBlock(tempReductionArray, reducedArray, blockDim.x);
	}
}

double probOfFilterOut111(MultiQubit multiQubit, const int idQubit1, const int idQubit2, const int idQubit3)
{
	long long int numValuesToReduce = multiQubit.numAmps;
	int valuesPerCUDABlock, numCUDABlocks, sharedMemSize;
	double stateProb=0;
	int firstTime=1;
	int maxReducedPerLevel = REDUCE_SHARED_SIZE;

	while(numValuesToReduce>1){	
		if (numValuesToReduce<maxReducedPerLevel){
			// Need less than one CUDA block to reduce values
			valuesPerCUDABlock = numValuesToReduce;
			numCUDABlocks = 1;
		} else {
			// Use full CUDA blocks, with block size constrained by shared mem usage
			valuesPerCUDABlock = maxReducedPerLevel;
			numCUDABlocks = ceil((double)numValuesToReduce/valuesPerCUDABlock);
		}
		sharedMemSize = valuesPerCUDABlock*sizeof(double);

		if (firstTime){
			probOfFilterOut111Kernel<<<numCUDABlocks, valuesPerCUDABlock, sharedMemSize>>>(
				multiQubit, idQubit1, idQubit2, idQubit3, multiQubit.firstLevelReduction);
			firstTime=0;
		} else {
			hipDeviceSynchronize();	
			copySharedReduceBlock<<<numCUDABlocks, valuesPerCUDABlock/2, sharedMemSize>>>(
				multiQubit.firstLevelReduction, 
				multiQubit.secondLevelReduction, valuesPerCUDABlock); 
			hipDeviceSynchronize();	
			swapDouble(&(multiQubit.firstLevelReduction), &(multiQubit.secondLevelReduction));
		}
		numValuesToReduce = numValuesToReduce/maxReducedPerLevel;
	}
	hipMemcpy(&stateProb, multiQubit.firstLevelReduction, sizeof(double), hipMemcpyDeviceToHost);
	return stateProb;

}



